#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <matrix.cuh>

#include <hittable_list.cuh>
#include <rt_weekend.cuh>

#include <interval.cuh>
#include <error_check.cuh>
#include <camera.cuh>

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// reference https://docs.nvidia.com/cuda/archive/12.0.1/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
// section 7.34
__global__ void create_world()
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        int num_spheres = 3;

        sphere_lst = (Hittable**)malloc(num_spheres * sizeof(Hittable*));
        world = (Hittable*)malloc(sizeof(Hittable*));

        metal = new Metal(Color(184.0/225.0, 115.0/225.0, 51.0/225.0), 0);
        lambertian = new Lambertian(Color(0.8, 0.8, 0.8));
        dielectric = new Dielectric(1.0 / 1.33);

        // printf("The memory address of metal is: %p\n", (void*)&metal);

        sphere_lst[0] = new Sphere(Point(0,-100.5,-1.5), 100, lambertian);
        sphere_lst[1] = new Sphere(Point(-0.51, 0, -1.5), 0.5, metal);
        sphere_lst[2] = new Sphere(Point(0.51, 0, -1.5), 0.5, dielectric);

        // printf("The memory address of sphere1 and sphere2 are: %p, %p\n", (void*)sphere_lst[0], (void*)sphere_lst[1]);
        // printf("Memory address of world: %p\n", (void*)world);

        world = new HittableList(sphere_lst, num_spheres);
    }

}

__global__ void free_world()
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        free(sphere_lst);
    }
}

__global__ void write_img(Matrix d_img, Camera camera, int samples_per_pixel, hiprandState* rand_states)
{
    SceneInfo scene_info = camera.get_scene_info();

    Vector pixel00_loc = scene_info.pixel00_loc;
    Vector camera_center = scene_info.camera_center;
    Vector pixel_delta_u = scene_info.pixel_delta_u;
    Vector pixel_delta_v = scene_info.pixel_delta_v;

    double pixel_samples_scale = 1.0 / samples_per_pixel;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    const Interval intensity(0.000, 0.999);

    if (row < d_img.height && col < d_img.width)
    {
        auto pixel_center = pixel00_loc + (col * pixel_delta_u) + (row * pixel_delta_v);
        auto ray_direction = pixel_center - camera_center;

        Color pixel_color(0, 0, 0);
        for (int sample = 0; sample < samples_per_pixel; sample++){
            Ray ray = camera.get_ray(col, row, rand_states);
            pixel_color += camera.ray_color(rand_states, 50, ray);
        }

        pixel_color = pixel_samples_scale * pixel_color;

        double r = pixel_color.x();
        double g = pixel_color.y();
        double b = pixel_color.z();

        d_img.at(row, col).x = static_cast<unsigned char>(255.999 * intensity.clamp(r));
        d_img.at(row, col).y = static_cast<unsigned char>(255.999 * intensity.clamp(g));
        d_img.at(row, col).z = static_cast<unsigned char>(255.999 * intensity.clamp(b));
    }
}

int main()
{
    // initialize constants empty and universal
    Interval h_empty(+infinity, -infinity);
    Interval h_universe(-infinity, +infinity);
    hipMemcpyToSymbol(HIP_SYMBOL(empty), &h_empty, sizeof(Interval));
    hipMemcpyToSymbol(HIP_SYMBOL(universe), &h_universe, sizeof(Interval));

    hipDeviceSetLimit(hipLimitStackSize, 131070);

    int samples_per_pixel = 50; 

    auto R = cos(pi/4);

    Camera camera;
    
    camera.aspect_ratio = 16.0 / 9.0;
    camera.img_width = 1200;
    camera.vfov = 20;
    camera.lookfrom = Point(-2,2,1);
    camera.lookat   = Point(0,0,-1);
    camera.vup      = Vector(0,1,0);
    camera.defocus_angle = 10.0;
    camera.focus_dist    = 3.4;

    camera.initialize();

    const int block_size = 16;

    int img_height = camera.get_img_height();

    SceneInfo scene_info = camera.get_scene_info();

    size_t size = camera.img_width * img_height * sizeof(uchar3);

    Matrix d_img(camera.img_width, img_height), h_img(camera.img_width, img_height);
    
    h_img.data = new uchar3[size];

    GPU_ERR_CHECK(hipMalloc(&d_img.data, size));

    dim3 dim_block(block_size, block_size);
    dim3 dim_grid((camera.img_width + dim_block.x-1) / block_size , (img_height + dim_block.y-1) / block_size ); 

    hiprandState* rand_states;
    int N = 2048;
    int num_threads = 512;
    int num_blocks = (N + num_threads - 1) / num_threads;
    hipMalloc((void**)&rand_states, N * sizeof(hiprandState));

    setup_kernel<<<num_blocks, num_threads>>>(rand_states, time(0));

    create_world<<<1,1>>>();

    write_img<<<dim_grid, dim_block>>>(d_img, camera, samples_per_pixel, rand_states);

    GPU_ERR_CHECK(hipDeviceSynchronize());

    GPU_ERR_CHECK(hipMemcpy(h_img.data, d_img.data, size, hipMemcpyDeviceToHost));

    std::ofstream ofs("../output/output.ppm", std::ios::out | std::ios::binary);
    ofs << "P3\n" << camera.img_width << ' ' << img_height << "\n255\n";
    for (int i = 0; i < img_height; i++) {
        for (int j = 0; j < camera.img_width; j++) {
            ofs << (int)h_img.at(i, j).x << ' '
                      << (int)h_img.at(i, j).y << ' '
                      << (int)h_img.at(i, j).z << '\n';
        }
    }

    free_world<<<1,1>>>();

    hipFree(d_img.data);
    hipFree(rand_states);
    delete[] h_img.data;

    return 0;
}
