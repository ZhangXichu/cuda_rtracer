#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <matrix.cuh>

#include <hittable_list.cuh>
#include <rt_weekend.cuh>

#include <interval.cuh>
#include <error_check.cuh>
#include <camera.cuh>
#include <triangle.cuh>
#include <obj_loader.cuh>

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__device__ inline Point vtxf(const float* V, int i) { // fetch vertex
    return Point((double)V[3*i+0], (double)V[3*i+1], (double)V[3*i+2]);
}

// yaw rotation around Y
__device__ inline Point yaw_y(const Point& p, double c, double s) {
    double x =  p.x()*c + p.z()*s;
    double z = -p.x()*s + p.z()*c;
    return Point(x, p.y(), z);
}

// reference https://docs.nvidia.com/cuda/archive/12.0.1/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
// section 7.34
__global__ void create_world(hiprandState* rand_states,
                             const float* verts, int vcount,
                             const int3* faces, int fcount,
                             double scale,      // S
                             Point translate,   // T
                             double yaw_deg)    // rotation about Y in degrees
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // int num_spheres = 190;

        // obj_lst = (Hittable**)malloc(num_spheres * sizeof(Hittable*));
        // world = (Hittable*)malloc(sizeof(Hittable*));

        // Material* ground = new Lambertian(Color(0.5, 0.5, 0.5));
        // obj_lst[0] = new Sphere(Point(0,-1000,0), 1000, ground);  // ground

        // Material* material1 = new Dielectric(1.5);
        // obj_lst[1] = new Sphere(Point(0, 1, 0), 1, material1);

        // Material* material2 = new Lambertian(Color(0.4, 0.2, 0.1));
        // obj_lst[2] = new Sphere(Point(-4, 1, 0), 1, material2);

        // Material* material3 = new Metal(Color(184.0/225.0, 115.0/225.0, 51.0/225.0), 0);
        // obj_lst[3] = new Sphere(Point(4, 1, 0), 1, material3);

        // int index = 4;

        // for (int a = -7; a < 7; a+=1.5)
        // {
        //     for (int b = -7; b < 7; b+=1.5) 
        //     {
        //         auto choose_mat = random_double(rand_states);

        //         Point center(a + 7.5*random_double(rand_states), 0.2, b + 7.5*random_double(rand_states));

        //         if ((center - Point(4, 0.2, 0)).length() > 0.9) 
        //         {
        //             Material* material;

        //             if (choose_mat < 0.5) 
        //             {
        //                 auto albedo = random_vec(rand_states) * random_vec(rand_states);
        //                 auto center2 = center + Vector(0, random_double(rand_states, 0, 1), 0);
                        
        //                 material = new Lambertian(albedo);
        //                 // obj_lst[index] = new Sphere(center, center2, 0.2, material);
        //                 obj_lst[index] = new Sphere(center, 0.2, material);
        //                 index += 1;
        //             } else if (choose_mat < 0.8) 
        //             {
        //                 auto albedo = random_vec(rand_states, 0.5, 1);
        //                 auto fuzz = random_double(rand_states, 0, 0.5);

        //                 material = new Metal(albedo, fuzz);
        //                 obj_lst[index] = new Sphere(center, 0.2, material);
        //                 index += 1;
                        
        //             } else {
        //                 material = new Dielectric(1.5);
        //                 obj_lst[index] = new Sphere(center, 0.2, material);
        //                 index += 1;
        //             }
        //         }

        //         if (index >= num_spheres - 1) {
        //             break;
        //         }
        //     }
        //     if (index >= num_spheres - 1) {
        //             break;
        //         }

        // }


        // world = new HittableList(obj_lst, index);



        ////////////// Triangle Example //////////////

        // int index = 1;

        // int num_obj = 2;

        // obj_lst = (Hittable**)malloc(num_obj * sizeof(Hittable*));
        
        // obj_lst[0] = new Sphere(Point(0,-1000,0), 1000, new Lambertian(Color(0.5,0.5,0.5)));

        // Material* tri = new Lambertian(Color(0.9, 0.1, 0.1));
        // Point v0(-0.8,  -0.4, -1.5);
        // Point v1( 0.8,  -0.4, -1.5);
        // Point v2( 0.0,   0.8, -1.5);
        // obj_lst[index++] = new Triangle(v0, v1, v2, tri);

        // world = new HittableList(obj_lst, index);

        ////////////// Triangle Example //////////////

        // Reserve: ground + all triangles
        const int num_max = 1 + fcount;
        obj_lst = (Hittable**)malloc(num_max * sizeof(Hittable*));

        // Ground
        obj_lst[0] = new Sphere(Point(0,-1000,0), 1000, new Lambertian(Color(0.5,0.5,0.5)));
        int index = 1;

        // Place/rotate params
        const double rad = yaw_deg * (pi/180.0);
        const double c = cos(rad), s = sin(rad);

        Material* tea_mat = new Lambertian(Color(0.75, 0.72, 0.68)); // clay-ish

        // Build triangles
        for (int f = 0; f < fcount && index < num_max; ++f) {
            int3 tri = faces[f];
            if (tri.x < 0 || tri.y < 0 || tri.z < 0 ||
                tri.x >= vcount || tri.y >= vcount || tri.z >= vcount) continue;

            Point a = vtxf(verts, tri.x);
            Point b = vtxf(verts, tri.y);
            Point c0 = vtxf(verts, tri.z);

            // scale
            a = Point(a.x()*scale,  a.y()*scale,  a.z()*scale);
            b = Point(b.x()*scale,  b.y()*scale,  b.z()*scale);
            c0 = Point(c0.x()*scale, c0.y()*scale, c0.z()*scale);

            // yaw
            a = yaw_y(a, c, s);
            b = yaw_y(b, c, s);
            c0 = yaw_y(c0,c, s);

            // translate
            a = a + translate;
            b = b + translate;
            c0 = c0+ translate;

            obj_lst[index++] = new Triangle(a, b, c0, tea_mat);
        }

        world = new HittableList(obj_lst, index);
    }

}

__global__ void free_world()
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        free(obj_lst);
    }
}

__global__ void write_img(Matrix d_img, Camera camera, int samples_per_pixel, hiprandState* rand_states)
{
    SceneInfo scene_info = camera.get_scene_info();

    Vector pixel00_loc = scene_info.pixel00_loc;
    Vector camera_center = scene_info.camera_center;
    Vector pixel_delta_u = scene_info.pixel_delta_u;
    Vector pixel_delta_v = scene_info.pixel_delta_v;

    double pixel_samples_scale = 1.0 / samples_per_pixel;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    const Interval intensity(0.000, 0.999);

    if (row < d_img.height && col < d_img.width)
    {
        auto pixel_center = pixel00_loc + (col * pixel_delta_u) + (row * pixel_delta_v);
        auto ray_direction = pixel_center - camera_center;

        Color pixel_color(0, 0, 0);
        for (int sample = 0; sample < samples_per_pixel; sample++){
            Ray ray = camera.get_ray(col, row, rand_states);
            pixel_color += camera.ray_color(rand_states, 50, ray);
        }

        pixel_color = pixel_samples_scale * pixel_color;

        double r = pixel_color.x();
        double g = pixel_color.y();
        double b = pixel_color.z();

        d_img.at(row, col).x = static_cast<unsigned char>(255.999 * intensity.clamp(r));
        d_img.at(row, col).y = static_cast<unsigned char>(255.999 * intensity.clamp(g));
        d_img.at(row, col).z = static_cast<unsigned char>(255.999 * intensity.clamp(b));
    }
}

int main()
{
    // initialize constants empty and universal
    Interval h_empty(+infinity, -infinity);
    Interval h_universe(-infinity, +infinity);
    hipMemcpyToSymbol(HIP_SYMBOL(empty), &h_empty, sizeof(Interval));
    hipMemcpyToSymbol(HIP_SYMBOL(universe), &h_universe, sizeof(Interval));

    hipDeviceSetLimit(hipLimitStackSize, 131070);

    size_t heapBytes = 256 * 1024 * 1024; // 256 MB
    hipDeviceSetLimit(hipLimitMallocHeapSize, heapBytes);

    int samples_per_pixel = 50; 

    auto R = cos(pi/4);

    Camera camera;
    
    // camera.aspect_ratio = 16.0 / 9.0;
    // camera.img_width = 1200;
    // camera.vfov = 23;  
    // camera.lookfrom = Point(13,3,5);
    // camera.lookat   = Point(0,0,0);
    // camera.vup      = Vector(0,1,0);
    // camera.defocus_angle = 0.1;
    // camera.focus_dist    = 3.0;

    ///////////// for teapot ///////////////
    camera.aspect_ratio = 16.0 / 9.0;
    camera.img_width = 1200;
    camera.vfov = 15.0;  
    camera.lookfrom = Point(8.0, 2.5, 3.0);
    camera.lookat   = Point(4.0, 1.25, 0.0);
    camera.vup      = Vector(0.0, 1.0, 0.0);
    camera.defocus_angle = 0.0;

    const double dx = camera.lookfrom.x() - camera.lookat.x();
    const double dy = camera.lookfrom.y() - camera.lookat.y();
    const double dz = camera.lookfrom.z() - camera.lookat.z();
    camera.focus_dist    = std::sqrt(dx*dx + dy*dy + dz*dz);
    ///////////// for teapot ///////////////

    camera.initialize();

    const int block_size = 16;

    int img_height = camera.get_img_height();

    SceneInfo scene_info = camera.get_scene_info();

    size_t size = camera.img_width * img_height * sizeof(uchar3);

    Matrix d_img(camera.img_width, img_height), h_img(camera.img_width, img_height);
    
    h_img.data = new uchar3[size];

    GPU_ERR_CHECK(hipMalloc(&d_img.data, size));

    dim3 dim_block(block_size, block_size);
    dim3 dim_grid((camera.img_width + dim_block.x-1) / block_size , (img_height + dim_block.y-1) / block_size ); 

    hiprandState* rand_states;
    int N = 2048;
    int num_threads = 512;
    int num_blocks = (N + num_threads - 1) / num_threads;
    hipMalloc((void**)&rand_states, N * sizeof(hiprandState));

    setup_kernel<<<num_blocks, num_threads>>>(rand_states, time(0));

    /////////////// teapot ///////////////
    // --- Load OBJ on host ---
    ObjLoader teapot;
    if (!teapot.load("/mnt/workspace/obj/teapot.obj")) {  // <- adjust path
        std::cerr << "Failed to load teapot.obj\n";
        return 1;
    }
    teapot.normalize();

    // Upload to device (Unified Memory)
    float* d_vertices = nullptr;
    int3*  d_faces    = nullptr;
    int    vcount = 0, fcount = 0;
    teapot.upload_to_device(d_vertices, vcount, d_faces, fcount);

    // Optional placement: size & pose in your scene
    double S = 2.5;                      // try 2–3
    Point  T(0.0, 0.4, -3.0);            // lift + push forward
    double yaw_deg = 0.0;                // try 90.0 if facing sideways

    // --- Build world with teapot ---
    create_world<<<1,1>>>(rand_states, d_vertices, vcount, d_faces, fcount, S, T, yaw_deg);
    
    GPU_ERR_CHECK(hipGetLastError());
    GPU_ERR_CHECK(hipDeviceSynchronize());
    /////////////// teapot ///////////////

    // create_world<<<1,1>>>(rand_states);

    write_img<<<dim_grid, dim_block>>>(d_img, camera, samples_per_pixel, rand_states);

    GPU_ERR_CHECK(hipDeviceSynchronize());
    GPU_ERR_CHECK(hipMemcpy(h_img.data, d_img.data, size, hipMemcpyDeviceToHost));

    std::ofstream ofs("../output/output.ppm", std::ios::out | std::ios::binary);
    ofs << "P3\n" << camera.img_width << ' ' << img_height << "\n255\n";
    for (int i = 0; i < img_height; i++) {
        for (int j = 0; j < camera.img_width; j++) {
            ofs << (int)h_img.at(i, j).x << ' '
                      << (int)h_img.at(i, j).y << ' '
                      << (int)h_img.at(i, j).z << '\n';
        }
    }

    free_world<<<1,1>>>();

    hipFree(d_img.data);
    hipFree(rand_states);
    hipFree(d_vertices);
    hipFree(d_faces);
    delete[] h_img.data;

    return 0;
}
